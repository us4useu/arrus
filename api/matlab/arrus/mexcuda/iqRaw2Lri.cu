#include "hip/hip_runtime.h"
#define M_PI 3.14159265358979
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <string>
#include <iostream>

__constant__ float xElemConst[1024];

texture <float2, hipTextureType1DLayered, hipReadModeElementType> iqRawTex;


__forceinline__ __device__ float ownHypotf(float x, float y)
{
    return sqrtf(x*x + y*y);
}


__global__ void iqRaw2Lri(  float2 * iqLri, float const * zPix, float const * xPix, 
                            float const * txFoc, float const * txAng, float const * txApCent, 
                            float const minRxTang, float const maxRxTang, 
                            float const fs, float const fn, 
                            float const sos, float const initDel, 
                            int const nZPix, int const nXPix, 
                            int const nSamp, int const nElem, 
                            int const nTx)
{
    int z = blockIdx.x * blockDim.x + threadIdx.x;
    int x = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (z>=nZPix || x>=nXPix) {
        return;
    }
    
    float txDist, rxDist, txTang, rxTang, txApod, rxApod, time, iSamp;
    float modSin, modCos, sampRe, sampIm, pixRe, pixIm, pixWgh;
    float const omega = 2 * M_PI * fn;
    float const sosInv = 1 / sos;
    float const zDistInv = 1 / zPix[z];
    float const nSigma = 3; // number of sigmas in half of the apodization Gaussian curve
    float const twoSigSqrInv = nSigma * nSigma * 0.5f;
    float const rngRxTangInv = 2 / (maxRxTang - minRxTang); // inverted half range
    float const centRxTang = (maxRxTang + minRxTang) * 0.5f;
    
    for (int iTx=0; iTx<nTx; iTx++) {
        
        if (!isinf(txFoc[iTx])) {
            /* STA */
            float xFoc	= txFoc[iTx] * sinf(txAng[iTx]) + txApCent[iTx];
            float zFoc	= txFoc[iTx] * cosf(txAng[iTx]);
            float minTxTang = (xFoc - xElemConst[0      ]) / zFoc;
            float maxTxTang = (xFoc - xElemConst[nElem-1]) / zFoc;  // invalid tx aperture edges (temporary solution)!!!
            
            txDist	= ownHypotf(zPix[z] - zFoc, xPix[x] - xFoc);
            //txDist	= txDist * sign(zPix[z] - zFoc) + txFoc[iTx];          // WARNING: sign()=0 => invalid txDist value
            txTang	= (xPix[x] - xFoc) / (zPix[z] - zFoc);
            txApod	= (txTang >= minTxTang && txTang <= maxTxTang) ? 1.f : 0.f;
        }
        else {
            /* PWI */
            float r1 = (xPix[x]-xElemConst[0      ]) * cosf(txAng[iTx]) - zPix[z] * sinf(txAng[iTx]);
            float r2 = (xPix[x]-xElemConst[nElem-1]) * cosf(txAng[iTx]) - zPix[z] * sinf(txAng[iTx]);
            
            txDist = xPix[x] * sinf(txAng[iTx]) + zPix[z] * cosf(txAng[iTx]);
            txApod = (r1 >= 0.f && r2 <= 0.f) ? 1.f : 0.f;
        }
        
        pixRe = 0.f;
        pixIm = 0.f;
        pixWgh = 0.f;
        
        if (txApod != 0.f) {
            for (int iElem=0; iElem<nElem; iElem++) {
                rxDist = ownHypotf(zPix[z], xPix[x] - xElemConst[iElem]);   // +10us
                rxTang = (xPix[x] - xElemConst[iElem]) * zDistInv;          // 4us
                if (rxTang < minRxTang || rxTang > maxRxTang) continue;
                rxApod = (rxTang-centRxTang)*rngRxTangInv;
                rxApod = __expf(-rxApod*rxApod*twoSigSqrInv);
                
                time = (txDist + rxDist) * sosInv + initDel;
                iSamp = time * fs;
                if (iSamp<0.f || iSamp>static_cast<float>(nSamp-1)) continue;
                
                float2 iqSamp = tex1DLayered(iqRawTex, iSamp + 0.5f, iElem + iTx*nElem);
                sampRe = iqSamp.x;
                sampIm = iqSamp.y;
                
                __sincosf(omega * time, &modSin, &modCos);
                
                pixRe += (sampRe * modCos - sampIm * modSin) * rxApod; // 60-80us
                pixIm += (sampRe * modSin + sampIm * modCos) * rxApod;
                pixWgh += rxApod;
            }
        }
        
        iqLri[z + x*nZPix + iTx*nZPix*nXPix].x = pixRe / pixWgh * txApod;
        iqLri[z + x*nZPix + iTx*nZPix*nXPix].y = pixIm / pixWgh * txApod;
    }
}

__host__ void checkData(mxGPUArray const * const data, char const * const name, bool const isComplex, int const nDims, char const * const invalidInputMsgId)
{
    std::string invalidInputMsgTxt(name);
    
    if (mxGPUGetClassID(data) != mxSINGLE_CLASS) 
        invalidInputMsgTxt += std::string(" must be single.");
    
    else if (!isComplex && mxGPUGetComplexity(data)) 
        invalidInputMsgTxt += std::string(" must be real.");
    
    else if (isComplex && !mxGPUGetComplexity(data)) 
        invalidInputMsgTxt += std::string(" must be complex.");
    
    else if (nDims==1 && !( mxGPUGetNumberOfDimensions(data) == 1 || 
                           (mxGPUGetNumberOfDimensions(data) == 2 && mxGPUGetDimensions(data)[0] == 1))) 
        invalidInputMsgTxt += std::string(" must be at most 1D vector.");
    
    else if (nDims==2 && !(mxGPUGetNumberOfDimensions(data) <= 2)) 
        invalidInputMsgTxt += std::string(" must be at most 2D array.");
    
    else if (nDims==3 && !(mxGPUGetNumberOfDimensions(data) <= 3)) 
        invalidInputMsgTxt += std::string(" must be at most 3D array.");
    
    else
        return;
    
    std::cout << " " << std::endl; // This line prevents crash, no idea why?
    mexErrMsgIdAndTxt( invalidInputMsgId, invalidInputMsgTxt.c_str());
}

void mexFunction(int nlhs, mxArray * plhs[],
                 int nrhs, mxArray const * prhs[])
{
    /* Initialize the GPU API. */
    mxInitGPU();
    
    /* Declare the variables */
    mxGPUArray * iqLri;
    mxGPUArray const * iqRaw;
    mxGPUArray const * xElem;
    mxGPUArray const * zPix;
    mxGPUArray const * xPix;
    mxGPUArray const * foc;
    mxGPUArray const * ang;
    mxGPUArray const * cent;
    
    float2 * dev_iqLri;
    float2 const * dev_iqRaw;
    float const * dev_xElem;
    float const * dev_zPix;
    float const * dev_xPix;
    float const * dev_foc;
    float const * dev_ang;
    float const * dev_cent;
    
    float minRxTang;
    float maxRxTang;
    float fs;
    float fn;
    float sos;
    float initDel;
    
    int nSamp;
    int nElem;
    int nZPix;
    int nXPix;
    int nTx;
    
    dim3 const threadsPerBlock = {16, 16, 1};
    dim3 blocksPerGrid;
    int sharedPerBlock;
    
    char const * const invalidInputMsgId = "iqRaw2Lri:InvalidInput";
    char const * const invalidOutputMsgId = "iqRaw2Lri:InvalidOutput";
    
    /* Validate mex inputs/outputs */
    if (nrhs!=13) {
        mexErrMsgIdAndTxt( invalidInputMsgId, "13 inputs required");
    }
    
    if (nlhs>1) {
        mexErrMsgIdAndTxt( invalidOutputMsgId, "One output allowed");
    }
    
//     for (int i=7; i<13; i++) {
//         if (!mxIsSingle(prhs[i]) || mxIsComplex(prhs[i]) || mxGetNumberOfElements(prhs[i]) != 1) {
//             mexErrMsgIdAndTxt( invalidInputMsgId, "Last 6 inputs must be single, real scalars");
//         }
//     }
    
    
    /* Extract inputs from prhs */
    iqRaw = mxGPUCreateFromMxArray(prhs[0]);
    xElem = mxGPUCreateFromMxArray(prhs[1]);
    zPix  = mxGPUCreateFromMxArray(prhs[2]);
    xPix  = mxGPUCreateFromMxArray(prhs[3]);
    foc   = mxGPUCreateFromMxArray(prhs[4]);
    ang   = mxGPUCreateFromMxArray(prhs[5]);
    cent  = mxGPUCreateFromMxArray(prhs[6]);
    
    minRxTang = mxGetScalar(prhs[7]);
    maxRxTang = mxGetScalar(prhs[8]);
    fs    = mxGetScalar(prhs[9]);
    fn    = mxGetScalar(prhs[10]);
    sos   = mxGetScalar(prhs[11]);
    initDel	= mxGetScalar(prhs[12]);
    
    /* Validate inputs */
    checkData(iqRaw,     "iqRaw",     true,  3, invalidInputMsgId);
    checkData(xElem,     "xElem",     false, 1, invalidInputMsgId);
    checkData(zPix,      "zPix",      false, 1, invalidInputMsgId);
    checkData(xPix,      "xPix",      false, 1, invalidInputMsgId);
    checkData(foc,       "foc",       false, 1, invalidInputMsgId);
    checkData(ang,       "ang",       false, 1, invalidInputMsgId);
    checkData(cent,      "cent",      false, 1, invalidInputMsgId);
    
    if (mxGPUGetDimensions(iqRaw)[1] != mxGPUGetNumberOfElements(xElem)) {
        mexErrMsgIdAndTxt( invalidInputMsgId, "size(iqRaw,2) must be equal to length(xElem).");
    }
    
    /* Get some additional information */
    nSamp = mxGPUGetDimensions(iqRaw)[0];
    nElem = mxGPUGetNumberOfElements(xElem);
    nZPix = mxGPUGetNumberOfElements(zPix);
    nXPix = mxGPUGetNumberOfElements(xPix);
    if (mxGPUGetNumberOfDimensions(iqRaw)<3) {
        nTx = 1;
    }
    else {
        nTx   = mxGPUGetDimensions(iqRaw)[2];
    }
    
    sharedPerBlock = 0;
    blocksPerGrid = {(nZPix+threadsPerBlock.x-1)/threadsPerBlock.x, 
                     (nXPix+threadsPerBlock.y-1)/threadsPerBlock.y, 1};
    
    /* Create output mxGPUArray object */
    mwSize nDimOut = 3;
    mwSize dimOut[3] = {nZPix, nXPix, nTx};
    
    iqLri = mxGPUCreateGPUArray(nDimOut,
                                dimOut,
                                mxSINGLE_CLASS,
                                mxCOMPLEX,
                                MX_GPU_DO_NOT_INITIALIZE);
    
    /* Get pointers on the device */
    dev_iqLri = static_cast<float2 *>(mxGPUGetData(iqLri));
    dev_iqRaw = static_cast<float2 const *>(mxGPUGetDataReadOnly(iqRaw));
    dev_xElem = static_cast<float const *>(mxGPUGetDataReadOnly(xElem));
    dev_zPix  = static_cast<float const *>(mxGPUGetDataReadOnly(zPix));
    dev_xPix  = static_cast<float const *>(mxGPUGetDataReadOnly(xPix));
    dev_foc   = static_cast<float const *>(mxGPUGetDataReadOnly(foc));
    dev_ang   = static_cast<float const *>(mxGPUGetDataReadOnly(ang));
    dev_cent  = static_cast<float const *>(mxGPUGetDataReadOnly(cent));
    
    /* set constant memory */
    if(nElem > 1024) {
        mexErrMsgIdAndTxt(invalidInputMsgId, "xElem is too long, kernel supports xElem of up to 1024 elements");
    }
    hipMemcpyToSymbol(HIP_SYMBOL(xElemConst), dev_xElem, nElem*sizeof(float), 0, hipMemcpyDeviceToDevice);
    
    /* configure texture reference */
    iqRawTex.normalized  = false;
    iqRawTex.addressMode[0] = hipAddressModeBorder;
    iqRawTex.filterMode  = hipFilterModeLinear;
    
    int nTxPerPart = (nElem*nTx <= 2048) ? nTx : 2048/nElem;
    int nPart = (nTx+nTxPerPart-1)/nTxPerPart;
    
    hipArray* cuArray;
    hipExtent cuArraySize =  make_hipExtent(nSamp, 0, nElem*nTxPerPart);
    hipMalloc3DArray(&cuArray, &iqRawTex.channelDesc, cuArraySize, hipArrayLayered);
    hipBindTextureToArray(iqRawTex, cuArray);
    
    /* Kernel in loop - due to limited number of texture layers */
    hipMemcpy3DParms cuArrayCopy = {0};
    cuArrayCopy.dstArray = cuArray;
    cuArrayCopy.kind = hipMemcpyDeviceToDevice;
    for (int iPart=0; iPart<nPart; iPart++) {
        
        int nTxInThisPart = (iPart<(nPart-1)) ? nTxPerPart : (nTx-iPart*nTxPerPart);
        
        /* Prepare texture memory */
        cuArrayCopy.srcPtr = make_hipPitchedPtr(const_cast<float2 *>(dev_iqRaw)+iPart*nSamp*nElem*nTxPerPart, nSamp * sizeof(float2), nSamp, 1);
        cuArrayCopy.extent = make_hipExtent(nSamp, 1, nElem*nTxInThisPart);
        hipMemcpy3D(&cuArrayCopy);
        
        /* Execute CUDA kernel */
        iqRaw2Lri<<<blocksPerGrid, threadsPerBlock, sharedPerBlock>>>(dev_iqLri + iPart*nZPix*nXPix*nTxPerPart, 
                                                                      dev_zPix, dev_xPix, 
                                                                      dev_foc       + iPart*nTxPerPart, 
                                                                      dev_ang       + iPart*nTxPerPart, 
                                                                      dev_cent      + iPart*nTxPerPart, 
                                                                      minRxTang, maxRxTang, fs, fn, sos, initDel, 
                                                                      nZPix, nXPix, nSamp, nElem, nTxInThisPart);
        
    }
    
    /* Wrap the output */
    plhs[0] = mxGPUCreateMxArrayOnGPU(iqLri);
    
    /* Clean-up */
    hipUnbindTexture(iqRawTex);
    hipFreeArray(cuArray);
    
    mxGPUDestroyGPUArray(iqLri);
    mxGPUDestroyGPUArray(iqRaw);
    mxGPUDestroyGPUArray(xElem);
    mxGPUDestroyGPUArray(zPix);
    mxGPUDestroyGPUArray(xPix);
    mxGPUDestroyGPUArray(foc);
    mxGPUDestroyGPUArray(ang);
    mxGPUDestroyGPUArray(cent);
    
    //hipDeviceReset();
}
