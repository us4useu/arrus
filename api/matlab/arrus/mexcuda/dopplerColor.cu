#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"

__global__ void dopplerColor(float * color, 
                             float * power, 
                             float2 const * iqImg, 
                             int const nZPix, 
                             int const nXPix, 
                             int const nRep)
{
    int z = blockIdx.x * blockDim.x + threadIdx.x;
    int x = blockIdx.y * blockDim.y + threadIdx.y;
    
    float2 iqPixCurr, iqPixPrev;
    float auxPower;
    float2 auxColor = {0.f, 0.f};
    
    if (z>=nZPix || x>=nXPix) {
        return;
    }
    
    /* Color & Power estimation */
    iqPixCurr = iqImg[z + x * nZPix];
    auxPower = iqPixCurr.x * iqPixCurr.x + iqPixCurr.y * iqPixCurr.y;
    for (int iRep=1; iRep<nRep; iRep++) {
        iqPixPrev = iqPixCurr;
        iqPixCurr = iqImg[z + x * nZPix + iRep * nZPix * nXPix];
        
        auxPower += iqPixCurr.x * iqPixCurr.x + iqPixCurr.y * iqPixCurr.y;
        auxColor.x += iqPixCurr.x * iqPixPrev.x + iqPixCurr.y * iqPixPrev.y;
        auxColor.y += iqPixCurr.y * iqPixPrev.x - iqPixCurr.x * iqPixPrev.y;
    }
    color[z + x*nZPix] = atan2f(auxColor.y, auxColor.x);
    power[z + x*nZPix] = auxPower / nRep;
}



void mexFunction(int nlhs, mxArray * plhs[],
                 int nrhs, mxArray const * prhs[])
{
    /* Initialize the GPU API. */
    mxInitGPU();
    
    /* Declare the variables */
    mxGPUArray * color;
    mxGPUArray * power;
    mxGPUArray const * iqImg;
    
    float * dev_color;
    float * dev_power;
    float2 const * dev_iqImg;
    
    int nZPix;
    int nXPix;
    int nRep;
    
    dim3 const threadsPerBlock = {32, 32, 1};
    dim3 blocksPerGrid;
    
    char const * const invalidInputMsgId = "dopplerColor:InvalidInput";
    char const * const invalidOutputMsgId = "dopplerColor:InvalidOutput";
    
    /* Validate mex inputs/outputs */
    if (nrhs!=1) {
        mexErrMsgIdAndTxt(invalidInputMsgId, "One input required");
    }
    
    if (nlhs>2) {
        mexErrMsgIdAndTxt(invalidOutputMsgId, "Two outputs allowed");
    }
    
    if (!(mxIsGPUArray(prhs[0]))) {
        mexErrMsgIdAndTxt(invalidInputMsgId, "Input must be gpuArray object containing single, complex 3D array");
    }
    
    /* Extract inputs from prhs */
    iqImg = mxGPUCreateFromMxArray(prhs[0]);
    
    /* Validate inputs */
    if( mxGPUGetClassID(iqImg) != mxSINGLE_CLASS || 
       !mxGPUGetComplexity(iqImg) || 
        mxGPUGetNumberOfDimensions(iqImg) != 3) {
        mexErrMsgIdAndTxt(invalidInputMsgId, "Input must be gpuArray object containing single, complex 3D array");
    }
    
    /* Get some additional information */
    nZPix = mxGPUGetDimensions(iqImg)[0];
    nXPix = mxGPUGetDimensions(iqImg)[1];
    nRep = mxGPUGetDimensions(iqImg)[2];
    
    blocksPerGrid = {(unsigned int)ceilf((float)nZPix/(float)threadsPerBlock.x), 
                     (unsigned int)ceilf((float)nXPix/(float)threadsPerBlock.y), 1};
    
    /* Create output mxGPUArray object */
    mwSize nDimOut = 2;
    mwSize dimOut[2] = {nZPix, nXPix};
    
    color = mxGPUCreateGPUArray(nDimOut, dimOut, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    power = mxGPUCreateGPUArray(nDimOut, dimOut, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    
    /* Get pointers on the device */
    dev_color = (float *)(mxGPUGetData(color));
    dev_power = (float *)(mxGPUGetData(power));
    dev_iqImg = (float2 const *)(mxGPUGetDataReadOnly(iqImg));
    
    /* Execute CUDA kernel */
    dopplerColor<<<blocksPerGrid, threadsPerBlock>>>(dev_color, dev_power, dev_iqImg, nZPix, nXPix, nRep);
    
    /* Wrap the output */
    plhs[0] = mxGPUCreateMxArrayOnGPU(color);
    plhs[1] = mxGPUCreateMxArrayOnGPU(power);
    
    /* Destroy the mxGPUArray objects */
    mxGPUDestroyGPUArray(color);
    mxGPUDestroyGPUArray(power);
    mxGPUDestroyGPUArray(iqImg);
    
}
