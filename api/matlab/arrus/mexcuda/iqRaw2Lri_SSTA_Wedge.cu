#include "hip/hip_runtime.h"
#define M_PI 3.14159265358979
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <string>
#include <iostream>

__constant__ float zElemConst[256];
__constant__ float xElemConst[256];
__constant__ float tangElemConst[256];

texture <float2, hipTextureType1DLayered, hipReadModeElementType> iqRawTex;


__forceinline__ __device__ float ownHypotf(float x, float y)
{
    return sqrtf(x*x + y*y);
}


__device__ float xRefract(	float const zElem, 
                            float const xElem, 
                            float const zPix, 
                            float const xPix, 
                            float const sosInterf, 
                            float const sosSample,
                            float const timePrec)
{
    float xRefractLo, sinRatioLo, xRefractHi, sinRatioHi, timeOld;
    float xRefractNew, distInterf, distSample, sinRatioNew, timeNew;
	
    float const cRatio = sosInterf / sosSample;
    
    // Initial refraction points
    xRefractLo = xElem;
    sinRatioLo = 0.f;
    
    xRefractHi = xElem - zElem * (xPix - xElem) / (zPix - zElem);
    sinRatioHi = 1.f;
    
    timeOld = ownHypotf(xRefractHi - xElem, zElem) / sosInterf
            + ownHypotf(xPix - xRefractHi, zPix ) / sosSample;
    
    // Iterations
    do {
        xRefractNew = xRefractLo + (xRefractHi-xRefractLo)*(cRatio-sinRatioLo)/(sinRatioHi-sinRatioLo);
        distInterf  = ownHypotf(xRefractNew - xElem, zElem);
        distSample  = ownHypotf(xPix - xRefractNew, zPix);
        sinRatioNew = ((xRefractNew - xElem) / distInterf) 
                    / ((xPix - xRefractNew) / distSample);
        timeNew     = distInterf / sosInterf + distSample / sosSample;
        
        if (fabs(timeNew-timeOld) < timePrec) {
            break;
        }
        
        if (sinRatioNew < cRatio) {
            xRefractLo = xRefractNew;
            sinRatioLo = sinRatioNew;
        }
        else {
            xRefractHi = xRefractNew;
            sinRatioHi = sinRatioNew;
        }
        timeOld = timeNew;
    } while(true);
    
    return xRefractNew;
}


__global__ void iqRaw2Lri(  float2 * iqLri, float const * zPix, float const * xPix, 
                            float const * txApCentZ, float const * txApCentX, 
                            int const * rxApOrigElem, 
                            float const minRxTang, float const maxRxTang, 
                            float const fs, float const fn, 
                            float const sos, 
                            float const sosInterf, 
                            float const timePrec, 
                            float const initDel, 
                            int const nZPix, int const nXPix, 
                            int const nSamp, int const nElem, 
                            int const nRx, 
                            int const nTx)
{
    int z = blockIdx.x * blockDim.x + threadIdx.x;
    int x = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (z>=nZPix || x>=nXPix) {
        return;
    }
    
    int iElem;
    float xRefr, txTime, rxTime, rxTang, txApod, rxApod, time, iSamp;
    float modSin, modCos, sampRe, sampIm, pixRe, pixIm, pixWgh;
    float const omega = 2 * M_PI * fn;
    float const sosInv = 1 / sos;
    float const sosInvInterf = 1 / sosInterf;
    float const nSigma = 3; // number of sigmas in half of the apodization Gaussian curve
    float const twoSigSqrInv = nSigma * nSigma * 0.5f;
    float const rngRxTangInv = 2 / (maxRxTang - minRxTang); // inverted half range
    float const centRxTang = (maxRxTang + minRxTang) * 0.5f;
    
    for (int iTx=0; iTx<nTx; iTx++) {
        
        /* SSTA */
        // Refraction point = [zRefr=0, xRefr]
        xRefr	= xRefract(txApCentZ[iTx], txApCentX[iTx], zPix[z], xPix[x], sosInterf, sos, timePrec);
        
        txTime	= ownHypotf(0.f - txApCentZ[iTx], xRefr - txApCentX[iTx]) * sosInvInterf
                + ownHypotf(zPix[z] - 0.f, xPix[x] - xRefr) * sosInv;
        
        txApod	= 1.f;  // temporary fixed
        
        pixRe = 0.f;
        pixIm = 0.f;
        pixWgh = 0.f;
        
        if (txApod != 0.f) {
            for (int iRx=0; iRx<nRx; iRx++) {
                iElem = iRx + rxApOrigElem[iTx];
                if (iElem<0 || iElem>=nElem) continue;
                
                xRefr	= xRefract(zElemConst[iElem], xElemConst[iElem], zPix[z], xPix[x], sosInterf, sos, timePrec);
                
                rxTime	= ownHypotf(xRefr - xElemConst[iElem], 0.f - zElemConst[iElem]) * sosInvInterf
                        + ownHypotf(xPix[x] - xRefr, zPix[z] - 0.f) * sosInv;
                
                rxTang = __fdividef(xRefr - xElemConst[iElem], 0.f - zElemConst[iElem]);
                rxTang = __fdividef(rxTang-tangElemConst[iElem], 1.f+rxTang*tangElemConst[iElem]);
                if (rxTang < minRxTang || rxTang > maxRxTang) continue;
                rxApod = (rxTang-centRxTang)*rngRxTangInv;
                rxApod = __expf(-rxApod*rxApod*twoSigSqrInv);
                
                time = txTime + rxTime + initDel;
                
                iSamp = time * fs;
                if (iSamp<0.f || iSamp>static_cast<float>(nSamp-1)) continue;
                
                float2 iqSamp = tex1DLayered(iqRawTex, iSamp + 0.5f, iRx + iTx*nRx);
                sampRe = iqSamp.x;
                sampIm = iqSamp.y;
                
                __sincosf(omega * time, &modSin, &modCos);
                
                pixRe += (sampRe * modCos - sampIm * modSin) * rxApod;
                pixIm += (sampRe * modSin + sampIm * modCos) * rxApod;
                pixWgh += rxApod;
            }
        }
        
        iqLri[z + x*nZPix + iTx*nZPix*nXPix].x = pixRe / pixWgh * txApod;
        iqLri[z + x*nZPix + iTx*nZPix*nXPix].y = pixIm / pixWgh * txApod;
    }
}

__host__ void checkData(mxGPUArray const * const data, char const * const name, bool const isComplex, int const nDims, char const * const invalidInputMsgId)
{
    std::string invalidInputMsgTxt(name);
    
    if (mxGPUGetClassID(data) != mxSINGLE_CLASS) 
        invalidInputMsgTxt += std::string(" must be single.");
    
    else if (!isComplex && mxGPUGetComplexity(data)) 
        invalidInputMsgTxt += std::string(" must be real.");
    
    else if (isComplex && !mxGPUGetComplexity(data)) 
        invalidInputMsgTxt += std::string(" must be complex.");
    
    else if (nDims==1 && !( mxGPUGetNumberOfDimensions(data) == 1 || 
                           (mxGPUGetNumberOfDimensions(data) == 2 && mxGPUGetDimensions(data)[0] == 1))) 
        invalidInputMsgTxt += std::string(" must be at most 1D vector.");
    
    else if (nDims==2 && !(mxGPUGetNumberOfDimensions(data) <= 2)) 
        invalidInputMsgTxt += std::string(" must be at most 2D array.");
    
    else if (nDims==3 && !(mxGPUGetNumberOfDimensions(data) <= 3)) 
        invalidInputMsgTxt += std::string(" must be at most 3D array.");
    
    else
        return;
    
    std::cout << " " << std::endl; // This line prevents crash, no idea why?
    mexErrMsgIdAndTxt( invalidInputMsgId, invalidInputMsgTxt.c_str());
}

void mexFunction(int nlhs, mxArray * plhs[],
                 int nrhs, mxArray const * prhs[])
{
    /* Initialize the GPU API. */
    mxInitGPU();
    
    /* Declare the variables */
    mxGPUArray * iqLri;
    mxGPUArray const * iqRaw;
    mxGPUArray const * zElem;
    mxGPUArray const * xElem;
    mxGPUArray const * tangElem;
    mxGPUArray const * zPix;
    mxGPUArray const * xPix;
//     mxGPUArray const * foc;
//     mxGPUArray const * ang;
    mxGPUArray const * centZ;
    mxGPUArray const * centX;
//     mxGPUArray const * elemFst;
//     mxGPUArray const * elemLst;
    mxGPUArray const * rxElemOrig;
    
    float2 * dev_iqLri;
    float2 const * dev_iqRaw;
    float const * dev_zElem;
    float const * dev_xElem;
    float const * dev_tangElem;
    float const * dev_zPix;
    float const * dev_xPix;
//     float const * dev_foc;
//     float const * dev_ang;
    float const * dev_centZ;
    float const * dev_centX;
//     int const * dev_elemFst;
//     int const * dev_elemLst;
    int const * dev_rxElemOrig;
    
    float minRxTang;
    float maxRxTang;
    float fs;
    float fn;
    float sos;
        float sosInterf;
        float timePrec;
    float initDel;
    
    int nSamp;
    int nElem;
    int nZPix;
    int nXPix;
    int nRx;
    int nTx;
    
    dim3 const threadsPerBlock = {16, 16, 1};
    dim3 blocksPerGrid;
    int sharedPerBlock;
    
    char const * const invalidInputMsgId = "iqRaw2Lri:InvalidInput";
    char const * const invalidOutputMsgId = "iqRaw2Lri:InvalidOutput";
    
    /* Validate mex inputs/outputs */
    if (nrhs!=17) {
        mexErrMsgIdAndTxt( invalidInputMsgId, "17 inputs required");
    }
    
    if (nlhs>1) {
        mexErrMsgIdAndTxt( invalidOutputMsgId, "One output allowed");
    }
    
//     for (int i=19; i<17; i++) {
//         if (!mxIsSingle(prhs[i]) || mxIsComplex(prhs[i]) || mxGetNumberOfElements(prhs[i]) != 1) {
//             mexErrMsgIdAndTxt( invalidInputMsgId, "Last 8 inputs must be single, real scalars");
//         }
//     }
    
    
    /* Extract inputs from prhs */
    iqRaw     = mxGPUCreateFromMxArray(prhs[0]);
    zElem     = mxGPUCreateFromMxArray(prhs[1]);
    xElem     = mxGPUCreateFromMxArray(prhs[2]);
    tangElem  = mxGPUCreateFromMxArray(prhs[3]);
    zPix      = mxGPUCreateFromMxArray(prhs[4]);
    xPix      = mxGPUCreateFromMxArray(prhs[5]);
//     foc       = mxGPUCreateFromMxArray(prhs[6]);
//     ang       = mxGPUCreateFromMxArray(prhs[7]);
    centZ     = mxGPUCreateFromMxArray(prhs[6]);
    centX     = mxGPUCreateFromMxArray(prhs[7]);
//     elemFst   = mxGPUCreateFromMxArray(prhs[10]);
//     elemLst   = mxGPUCreateFromMxArray(prhs[11]);
    rxElemOrig  = mxGPUCreateFromMxArray(prhs[8]);
    
    minRxTang = mxGetScalar(prhs[9]);
    maxRxTang = mxGetScalar(prhs[10]);
    fs        = mxGetScalar(prhs[11]);
    fn        = mxGetScalar(prhs[12]);
    sos       = mxGetScalar(prhs[13]);
        sosInterf = mxGetScalar(prhs[14]);
        timePrec  = mxGetScalar(prhs[15]);
    initDel   = mxGetScalar(prhs[16]);
    
    /* Validate inputs */
    checkData(iqRaw,     "iqRaw",     true,  3, invalidInputMsgId);
    checkData(zElem,     "zElem",     false, 1, invalidInputMsgId);
    checkData(xElem,     "xElem",     false, 1, invalidInputMsgId);
    checkData(tangElem,  "tangElem",  false, 1, invalidInputMsgId);
    checkData(zPix,      "zPix",      false, 1, invalidInputMsgId);
    checkData(xPix,      "xPix",      false, 1, invalidInputMsgId);
//     checkData(foc,       "foc",       false, 1, invalidInputMsgId);
//     checkData(ang,       "ang",       false, 1, invalidInputMsgId);
    checkData(centZ,     "centZ",     false, 1, invalidInputMsgId);
    checkData(centX,     "centX",     false, 1, invalidInputMsgId);
//     checkData(elemFst,   "elemFst",   false, 1, invalidInputMsgId); //int
//     checkData(elemLst,   "elemLst",   false, 1, invalidInputMsgId); //int
//     checkData(rxElemOrig,"rxElemOrig",false, 1, invalidInputMsgId); //int
    
    /* Get some additional information */
    nSamp = mxGPUGetDimensions(iqRaw)[0];
    nRx   = mxGPUGetDimensions(iqRaw)[1];
    nElem = mxGPUGetNumberOfElements(xElem);
    nZPix = mxGPUGetNumberOfElements(zPix);
    nXPix = mxGPUGetNumberOfElements(xPix);
    if (mxGPUGetNumberOfDimensions(iqRaw)<3) {
        nTx = 1;
    }
    else {
        nTx   = mxGPUGetDimensions(iqRaw)[2];
    }
    
    sharedPerBlock = 0;
    blocksPerGrid = {(nZPix+threadsPerBlock.x-1)/threadsPerBlock.x, 
                     (nXPix+threadsPerBlock.y-1)/threadsPerBlock.y, 1};
    
    /* Create output mxGPUArray object */
    mwSize nDimOut = 3;
    mwSize dimOut[3] = {nZPix, nXPix, nTx};
    
    iqLri = mxGPUCreateGPUArray(nDimOut,
                                dimOut,
                                mxSINGLE_CLASS,
                                mxCOMPLEX,
                                MX_GPU_DO_NOT_INITIALIZE);
    
    /* Get pointers on the device */
    dev_iqLri    = static_cast<float2 *>(mxGPUGetData(iqLri));
    dev_iqRaw    = static_cast<float2 const *>(mxGPUGetDataReadOnly(iqRaw));
    dev_zElem    = static_cast<float const *>(mxGPUGetDataReadOnly(zElem));
    dev_xElem    = static_cast<float const *>(mxGPUGetDataReadOnly(xElem));
    dev_tangElem = static_cast<float const *>(mxGPUGetDataReadOnly(tangElem));
    dev_zPix     = static_cast<float const *>(mxGPUGetDataReadOnly(zPix));
    dev_xPix     = static_cast<float const *>(mxGPUGetDataReadOnly(xPix));
//     dev_foc      = static_cast<float const *>(mxGPUGetDataReadOnly(foc));
//     dev_ang      = static_cast<float const *>(mxGPUGetDataReadOnly(ang));
    dev_centZ    = static_cast<float const *>(mxGPUGetDataReadOnly(centZ));
    dev_centX    = static_cast<float const *>(mxGPUGetDataReadOnly(centX));
//     dev_elemFst  = static_cast<int const *>(mxGPUGetDataReadOnly(elemFst));
//     dev_elemLst  = static_cast<int const *>(mxGPUGetDataReadOnly(elemLst));
    dev_rxElemOrig  = static_cast<int const *>(mxGPUGetDataReadOnly(rxElemOrig));
    
    /* set constant memory */
    if(nElem > 256) {
        mexErrMsgIdAndTxt(invalidInputMsgId, "xElem is too long, kernel supports xElem of up to 256 elements");
    }
    hipMemcpyToSymbol(HIP_SYMBOL(   zElemConst), dev_zElem,    nElem*sizeof(float), 0, hipMemcpyDeviceToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(   xElemConst), dev_xElem,    nElem*sizeof(float), 0, hipMemcpyDeviceToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(tangElemConst), dev_tangElem, nElem*sizeof(float), 0, hipMemcpyDeviceToDevice);
    
    /* configure texture reference */
    iqRawTex.normalized  = false;
    iqRawTex.addressMode[0] = hipAddressModeBorder;
    iqRawTex.filterMode  = hipFilterModeLinear;
    
    int nTxPerPart = (nRx*nTx <= 2048) ? nTx : 2048/nRx;
    int nPart = (nTx+nTxPerPart-1)/nTxPerPart;
    
    hipArray* cuArray;
    hipExtent cuArraySize =  make_hipExtent(nSamp, 0, nRx*nTxPerPart);
    hipMalloc3DArray(&cuArray, &iqRawTex.channelDesc, cuArraySize, hipArrayLayered);
    hipBindTextureToArray(iqRawTex, cuArray);
    
    /* Kernel in loop - due to limited number of texture layers */
    hipMemcpy3DParms cuArrayCopy = {0};
    cuArrayCopy.dstArray = cuArray;
    cuArrayCopy.kind = hipMemcpyDeviceToDevice;
    for (int iPart=0; iPart<nPart; iPart++) {
        
        int nTxInThisPart = (iPart<(nPart-1)) ? nTxPerPart : (nTx-iPart*nTxPerPart);
        
        /* Prepare texture memory */
        cuArrayCopy.srcPtr = make_hipPitchedPtr(const_cast<float2 *>(dev_iqRaw)+iPart*nSamp*nRx*nTxPerPart, nSamp * sizeof(float2), nSamp, 1);
        cuArrayCopy.extent = make_hipExtent(nSamp, 1, nRx*nTxInThisPart);
        hipMemcpy3D(&cuArrayCopy);
        
        /* Execute CUDA kernel */
//         iqRaw2Lri<<<blocksPerGrid, threadsPerBlock, sharedPerBlock>>>(dev_iqLri + iPart*nZPix*nXPix*nTxPerPart, 
//                                                                       dev_zPix, dev_xPix, 
//                                                                       dev_foc       + iPart*nTxPerPart, 
//                                                                       dev_ang       + iPart*nTxPerPart, 
//                                                                       dev_centZ     + iPart*nTxPerPart, 
//                                                                       dev_centX     + iPart*nTxPerPart, 
//                                                                       dev_elemFst   + iPart*nTxPerPart, 
//                                                                       dev_elemLst   + iPart*nTxPerPart, 
//                                                                       dev_rxElemOrig + iPart*nTxPerPart, 
//                                                                       minRxTang, maxRxTang, fs, fn, sos, initDel, 
//                                                                       nZPix, nXPix, nSamp, nElem, nRx, nTxInThisPart);
        iqRaw2Lri<<<blocksPerGrid, threadsPerBlock, sharedPerBlock>>>(dev_iqLri + iPart*nZPix*nXPix*nTxPerPart, 
                                                                      dev_zPix, dev_xPix, 
//                                                                       dev_foc       + iPart*nTxPerPart, 
//                                                                       dev_ang       + iPart*nTxPerPart, 
                                                                      dev_centZ     + iPart*nTxPerPart, 
                                                                      dev_centX     + iPart*nTxPerPart, 
//                                                                       dev_elemFst   + iPart*nTxPerPart, 
//                                                                       dev_elemLst   + iPart*nTxPerPart, 
                                                                      dev_rxElemOrig + iPart*nTxPerPart, 
                                                                      minRxTang, maxRxTang, fs, fn, sos, sosInterf, timePrec, initDel, 
                                                                      nZPix, nXPix, nSamp, nElem, nRx, nTxInThisPart);
        
    }
    
    /* Wrap the output */
    plhs[0] = mxGPUCreateMxArrayOnGPU(iqLri);
    
    /* Clean-up */
    hipUnbindTexture(iqRawTex);
    hipFreeArray(cuArray);
    
    mxGPUDestroyGPUArray(iqLri);
    mxGPUDestroyGPUArray(iqRaw);
    mxGPUDestroyGPUArray(zElem);
    mxGPUDestroyGPUArray(xElem);
    mxGPUDestroyGPUArray(tangElem);
    mxGPUDestroyGPUArray(zPix);
    mxGPUDestroyGPUArray(xPix);
//     mxGPUDestroyGPUArray(foc);
//     mxGPUDestroyGPUArray(ang);
    mxGPUDestroyGPUArray(centZ);
    mxGPUDestroyGPUArray(centX);
//     mxGPUDestroyGPUArray(elemFst);
//     mxGPUDestroyGPUArray(elemLst);
    mxGPUDestroyGPUArray(rxElemOrig);
    
    //hipDeviceReset();
}
